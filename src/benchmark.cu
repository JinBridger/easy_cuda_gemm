#include "hip/hip_runtime.h"
#include "benchmark.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include ""
#include "easy_cuda_gemm.h"

#include <chrono>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <omp.h>

float* generate_float_matrix(int size) {
    float* mat = new float[size * size];

#pragma omp parallel for
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            mat[i * size + j] = float(rand()) / RAND_MAX;
        }
    }

    return mat;
}

void generate_float_timer(unsigned int size) {
    auto start = std::chrono::system_clock::now();

    float* mat = generate_float_matrix(size);

    auto end      = std::chrono::system_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "Generate " << size << " matrix cost " << duration.count() << " ms" << std::endl;

    delete[] mat;
}

void matrix_differ(float* correct_val, float* test_val, unsigned int size) {
    float max_diff = 0.0f;

    for (unsigned int i = 0; i < size * size; ++i) {
        float diff = fabs(correct_val[i] - test_val[i]);
        max_diff   = std::max(max_diff, diff);
    }

    std::cout << "Max diff: " << max_diff << std::endl;
}

void gflops_benchmark(unsigned int size, bool is_baseline) {
    size_t size_of_matrix = sizeof(float) * size * size;

    float* host_mat_A = generate_float_matrix(size);
    float* host_mat_B = generate_float_matrix(size);

    float* device_mat_A;
    float* device_mat_B;
    float* device_mat_C;
    hipMalloc(( void** )&device_mat_A, size_of_matrix);
    hipMalloc(( void** )&device_mat_B, size_of_matrix);
    hipMalloc(( void** )&device_mat_C, size_of_matrix);

    hipMemcpy(( void* )device_mat_A, ( void* )host_mat_A, size_of_matrix, hipMemcpyHostToDevice);
    hipMemcpy(( void* )device_mat_B, ( void* )host_mat_B, size_of_matrix, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1;
    float beta  = 0;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);

    if (is_baseline) {
        for (int i = 0; i < benchmark_cycle; ++i) {
            hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, size, size, size, &alpha, device_mat_A, size, device_mat_B,
                        size, &beta, device_mat_C, size);
        }
    }
    else {
        for (int i = 0; i < benchmark_cycle; ++i) {
            easy_cuda_gemm(handle, size, size, size, device_mat_A, size, device_mat_B, size, device_mat_C, size);
        }
    }

    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, end);
    float avg_time = elapsed_time / benchmark_cycle;

    double flops_per_matmul = 2.0 * size * size * size;
    double avg_gflops       = (flops_per_matmul * 1.0e-9f) / (avg_time / 1000.0f);

    std::cout << "Size: " << size << "\t";
    std::cout << "GFLOPS: " << avg_gflops << "\t";
    std::cout << "Time: " << avg_time << std::endl;

    hipFree(( void* )device_mat_A);
    hipFree(( void* )device_mat_B);
    hipFree(( void* )device_mat_C);

    delete[] host_mat_A;
    delete[] host_mat_B;
}

void correctness_benchmark(unsigned int size) {
    size_t size_of_matrix = sizeof(float) * size * size;

    float* host_mat_A = generate_float_matrix(size);
    float* host_mat_B = generate_float_matrix(size);

    float* host_mat_correct = new float[size * size];
    float* host_mat_test    = new float[size * size];

    float* device_mat_A;
    float* device_mat_B;
    float* device_mat_C;
    hipMalloc(( void** )&device_mat_A, size_of_matrix);
    hipMalloc(( void** )&device_mat_B, size_of_matrix);
    hipMalloc(( void** )&device_mat_C, size_of_matrix);

    hipMemcpy(( void* )device_mat_A, ( void* )host_mat_A, size_of_matrix, hipMemcpyHostToDevice);
    hipMemcpy(( void* )device_mat_B, ( void* )host_mat_B, size_of_matrix, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1;
    float beta  = 0;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, device_mat_B, size, device_mat_A, size,
                &beta, device_mat_C, size);

    hipMemcpy(( void* )host_mat_correct, ( void* )device_mat_C, size_of_matrix, hipMemcpyDeviceToHost);

    easy_cuda_gemm(handle, size, size, size, device_mat_A, size, device_mat_B, size, device_mat_C, size);

    hipMemcpy(( void* )host_mat_test, ( void* )device_mat_C, size_of_matrix, hipMemcpyDeviceToHost);

    hipFree(( void* )device_mat_A);
    hipFree(( void* )device_mat_B);
    hipFree(( void* )device_mat_C);

    matrix_differ(host_mat_correct, host_mat_test, size);

    delete[] host_mat_A;
    delete[] host_mat_B;
    delete[] host_mat_correct;
    delete[] host_mat_test;
}